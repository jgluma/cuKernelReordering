/**
 * @file cublasrotm.cu
 * @details This file describes the functions belonging to CUBLASROTM class.
 * @author Antonio Jose Lazaro Munoz.
 * @date 20/02/2016
 */
#include "cublasrotm.h"


CUBLASROTM::CUBLASROTM(int s)
{
	h_x = NULL;
	h_y = NULL;
	d_x = NULL;
	d_y = NULL;
	h_H = NULL;
	d_H = NULL;

	size = s;

}

CUBLASROTM::~CUBLASROTM()
{
	//Free host memory
	if(h_x != NULL)   hipHostFree(h_x);
	if(h_y != NULL)   hipHostFree(h_y);
	if(h_H != NULL)   hipHostFree(h_H);

	//Free device memory
	if(d_x != NULL)	hipFree(d_x);
	if(d_y != NULL)	hipFree(d_y);	

}

void CUBLASROTM::allocHostMemory(void)
{
		
	
	hipHostMalloc((void **)&h_x, size * sizeof(float));
	hipHostMalloc((void **)&h_y, size * sizeof(float));
	hipHostMalloc((void **)&h_H, 5 * sizeof(float));
	
	
}

void CUBLASROTM::freeHostMemory(void)
{
	
	if(h_x != NULL)   hipHostFree(h_x);
	if(h_y != NULL)   hipHostFree(h_y);
	if(h_H != NULL)   hipHostFree(h_H);
	
	
}

void CUBLASROTM::allocDeviceMemory(void)
{
	
	hipMalloc((void **)&d_x, size * sizeof(float));
	hipMalloc((void **)&d_y, size * sizeof(float));
	
	
}

void CUBLASROTM::freeDeviceMemory(void)
{
	
	if(d_x != NULL)	hipFree(d_x);
	if(d_y != NULL)	hipFree(d_y);
	
}

void CUBLASROTM::generatingData(void)
{
	
	//Generating Data vectors
	srand(time(NULL));
	
	for(int i = 0; i < size; i++)
	{
		h_x[i] = (float)(1 + rand()%1000);
		h_y[i] = (float)(1 + rand()%1000);
	}
		
		
	
	
	
	h_H[0] = -1.0;
	for(int i = 0; i < 4; i++)
	{
		h_H[1 + i] = (float)(1 + rand()%40);
		
	}
	
	
	//Creating handle CUBLAS
	hipblasCreate(&handle);
	
}

void CUBLASROTM::memHostToDeviceAsync(hipStream_t stream)
{
	hipMemcpyAsync(d_x, h_x, sizeof(float)*size, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(d_y, h_y, sizeof(float)*size, hipMemcpyHostToDevice, stream);
}

void CUBLASROTM::memHostToDevice(void)
{
   	hipMemcpy(d_x, h_x, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, sizeof(float)*size, hipMemcpyHostToDevice);
}

void CUBLASROTM::memDeviceToHostAsync(hipStream_t stream)
{
	//Load the column indices to the gpu
	hipMemcpyAsync(h_x, d_x, sizeof(float)*size, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_y, d_y, sizeof(float)*size, hipMemcpyDeviceToHost, stream);
}

void CUBLASROTM::memDeviceToHost(void)
{

  //Load the column indices to the gpu
  hipMemcpyAsync(h_x, d_x, sizeof(float)*size, hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_y, d_y, sizeof(float)*size, hipMemcpyDeviceToHost);

}

void CUBLASROTM::launch_kernel_Async(hipStream_t stream)
{
	
	hipblasStatus_t status;
	 
	hipblasSetStream(handle, stream);
	
	status = hipblasSrotm(handle, size, d_x, incx, d_y, incy, h_H);
	if(status != HIPBLAS_STATUS_SUCCESS )
	{
		cout << "Error CUBLAS" << endl;
		exit(1);
		
	}
	
	
}

void CUBLASROTM::launch_kernel(void)
{

    hipblasStatus_t status;
	
	status = hipblasSrotm(handle, size, d_x, incx, d_y, incy, h_H);
	if(status != HIPBLAS_STATUS_SUCCESS )
	{
		cout << "Error CUBLAS" << endl;
		exit(1);
		
	}


}

void CUBLASROTM::checkResults(void)
{
	
	
}


void CUBLASROTM::getBytesHTD(int *bytes_htd)
{
	
	
	*bytes_htd = (sizeof(float)*size)*2;
	
	
}

void CUBLASROTM::getBytesDTH(int *bytes_dth)
{
	
	*bytes_dth = (sizeof(float)*size)*2;
	
	
}

void CUBLASROTM::getTimeEstimations_HTD_DTH(int gpu, float *estimated_time_HTD, float *estimated_time_DTH,
								float *estimated_overlapped_time_HTD, float *estimated_overlapped_time_DTH, 
								float LoHTD, float LoDTH, float GHTD, float GDTH, float overlappedGHTD, float overlappedGDTH)
{
	
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, gpu);

	int bytes_HTD;
	int bytes_DTH;

	getBytesHTD(&bytes_HTD);
	getBytesDTH(&bytes_DTH);
	
	
			
	*estimated_time_HTD = LoHTD + (bytes_HTD) * GHTD;
				
	*estimated_overlapped_time_HTD = 0.0;
		
	if(props.asyncEngineCount == 2)
		*estimated_overlapped_time_HTD = LoHTD + (bytes_HTD) * overlappedGHTD;
			
		
	*estimated_time_DTH = LoDTH + (bytes_DTH) * GDTH;
				
	*estimated_overlapped_time_DTH= 0.0;

		
	if(props.asyncEngineCount == 2)
		*estimated_overlapped_time_DTH= LoDTH + (bytes_DTH) * overlappedGDTH;

	
	
}
